//Implemented By Christopher Masloub
//Student Number: 20052223

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <memory>
#include <string>

void printDevProp(hipDeviceProp_t devProp)
{
	printf("Device Name:                          %s\n", devProp.name);
	printf("GPU Clock:                %.0f MHz (%0.2f ""GHz)\n", devProp.clockRate * 1e-3f, devProp.clockRate * 1e-6f);
	printf("Number of cuda cores:          %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor) * devProp.multiProcessorCount);
	printf("Number SMs:                    %d\n", devProp.multiProcessorCount);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
	printf("Total constant memory:         %lu\n", devProp.totalConstMem);
	printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);



}

int main()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	printf("CUDA Device Query\n");
	printf("CUDA devices : [%d] \n", devCount);

	for (int i = 0; i < devCount; ++i)
	{
		printf("\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printDevProp(devProp);

	}

	
}